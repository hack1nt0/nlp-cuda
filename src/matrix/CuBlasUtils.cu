//
// Created by DY on 17-11-10.
//

#ifndef NLP_CUDA_CUBLASUTILS_H
#define NLP_CUDA_CUBLASUTILS_H

#include "CuDenseMatrix.cu"
#include "CuSparseMatrix.cu"

struct CudaSparseContext {
    hipsparseHandle_t handle;

    CudaSparseContext() {
        hipsparseCreate(&handle);
        hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);
    }

    virtual ~CudaSparseContext() {
        hipsparseDestroy(handle);
    }
};
CudaSparseContext cudaSparseContext;

/*
 * C = alpha * A * B + beta * C
 * */
template <typename T>
void cuSparseMultiplyDense(CuDenseMatrix<T> &C, T beta,
                           T alpha, const CuSparseMatrix<T> &A, bool transposeA,
                           const CuDenseMatrix<T> &B, bool transposeB) {

    hipsparseOperation_t transA = transposeA ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t transB = transposeB ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;
    if (A.type == "d") {
        checkCusparseErrors(
            hipsparseDcsrmm2(cudaSparseContext.handle, //todo
                            transA,
                            transB,
                            A.rows,
                            B.cols,
                            A.cols,
                            A.nnz,
                            &alpha,
                            A.descr,
                            A.data,
                            A.row_ptr,
                            A.index,
                            B.data,
                            B.rows,
                            &beta,
                            C.data,
                            C.rows)
        );
    } else if (A.type == "f") {
        checkCusparseErrors(
            hipsparseScsrmm2(cudaSparseContext.handle, //todo
                            transA,
                            transB,
                            A.rows,
                            B.cols,
                            A.cols,
                            A.nnz,
                            &alpha,
                            A.descr,
                            A.data,
                            A.row_ptr,
                            A.index,
                            B.data,
                            B.rows,
                            &beta,
                            C.data,
                            C.rows)
        );
    }
}

struct CudaBlasContext {
    hipblasHandle_t handle;

    CudaBlasContext() {
        hipblasCreate(&handle);
        hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
    }

    virtual ~CudaBlasContext() {
        hipblasDestroy(handle);
    }
};
const CudaBlasContext cudaBlasContext;

#endif //NLP_CUDA_CUBLASUTILS_H
